/*
  mavrkons_apsp.cu

  All Pair Shortest Path (APSP)

  Project C

  Author: <Konstantinos Mavrodis>
  Contact: <kmavrodis@outlook.com>
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

void initialize(int *n, float *p, int *w, int argc, char const **argv);
void makeAdjacency(float *adjacency, int n, float p , int w);
void serialApsp(float *adjacency, float *distance, int n, float p , int w);
void cudaA(float *adjacency, float *cudaA_distance, int n, float p, int w);
void cudaB(float *adjacency, float *cudaB_distance, int n, float p, int w);
void cudaC(float *adjacency, float *cudaC_distance, int n, float p, int w);
__global__ void cudaA_kernel(float *d_cudaA_distance, int n, int k);
__global__ void cudaB_kernel(float *d_cudaB_distance, int n, int k);
__global__ void cudaC_kernel(float *d_cudaC_distance, int n, int k, int cellsPerThread);
int compare(float *arrayA, float *arrayB, int n);



int main(int argc, char const **argv)
{
  int n, w;
  float p, *adjacency, *distance, *cudaA_distance, *cudaB_distance, *cudaC_distance;
  double serialTime, cudaATime, cudaBTime, cudaCTime;
  struct timespec tstart={0,0}, tend={0,0};

  initialize(&n, &p, &w, argc, argv);
  
  adjacency = (float*) malloc(n*n * sizeof(float));
  distance = (float*) malloc(n*n * sizeof(float));
  cudaA_distance = (float*) malloc(n*n * sizeof(float));
  cudaB_distance = (float*) malloc(n*n * sizeof(float));
  cudaC_distance = (float*) malloc(n*n * sizeof(float));
  
  makeAdjacency(adjacency, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tstart);

  serialApsp(adjacency, distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  serialTime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) -
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);


  clock_gettime(CLOCK_MONOTONIC, &tstart);  

  cudaA(adjacency, cudaA_distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  cudaATime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);


  clock_gettime(CLOCK_MONOTONIC, &tstart);  

  cudaB(adjacency, cudaB_distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  cudaBTime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
  

  clock_gettime(CLOCK_MONOTONIC, &tstart);  

  cudaC(adjacency, cudaC_distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  cudaCTime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);


  printf("n=%i p=%g w=%i\n", n, p, w);
  printf("Serial time: %f\n", serialTime);
  printf("Cuda A time: %f\n", cudaATime);
  printf("Cuda B time: %f\n", cudaBTime);
  printf("Cuda C time: %f\n", cudaCTime);


  if(compare(cudaA_distance, distance,n) && compare(cudaB_distance, distance,n)
    && (compare(cudaC_distance, distance,n)))
    printf("PASS: All algorithms ran correctly.\n");
  else
    printf("FAIL: There was a wrong result.\n");

}


void initialize(int *n, float *p, int *w, int argc, char const **argv)
{
  if (argc == 4)
  {
    *n = 1<<atoi(argv[1]);
    *p = atof(argv[2]);
    *w = atoi(argv[3]); 
  }
  else
  {
    printf("The defaults for n,p,w (7,0.7,30) will be used\n");
    *n = 128;
    *p = 0.7;
    *w = 30;
  }
}

void makeAdjacency(float *adjacency, int n, float p , int w)
{
  int i, j;
  time_t t;
  srand((unsigned) time(&t));

  for (i = 0; i < n*n; ++i)
    adjacency[i] = 0;


  for (i = 0; i < n; ++i)
  {
    for (j = 0; j < n; ++j)
    {
      if ((float)(rand()%1000)/1000 > p)
        adjacency[i*n+j] = INFINITY;
      else
        adjacency[i*n+j] = ((float)(rand()%1000)/1000) * w;
    }
    adjacency[i*n+i] = 0;
  }
}

void serialApsp(float *adjacency, float *distance, int n, float p , int w)
{
  int i, j, k;

  for (i = 0; i < n; ++i)
    for (j = 0; j < n; ++j)
      distance[i*n+j] = adjacency[i*n+j];

  for (k = 0; k < n; ++k)
    for (i = 0; i < n; ++i)
      for (j = 0; j < n; ++j)
        if (distance[i*n+j] > distance[i*n+k] + distance[k*n+j])
          distance[i*n+j] = distance[i*n+k] + distance[k*n+j];
}

void cudaA(float *adjacency, float *cudaA_distance, int n, float p, int w)
{
  float *d_cudaA_distance;  
  hipMalloc(&d_cudaA_distance, n*n*sizeof(float));

  hipMemcpy(d_cudaA_distance, adjacency, n*n*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 threadsPerBlock; 
  dim3 blocksPerGrid;

  threadsPerBlock.x = threadsPerBlock.y = 32;
  blocksPerGrid.x = blocksPerGrid.y = n/32;

  int k;
  for (k = 0; k < n; ++k)
    cudaA_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_cudaA_distance, n, k);

  hipMemcpy(cudaA_distance, d_cudaA_distance, n*n*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaB(float *adjacency, float *cudaB_distance, int n, float p, int w)
{
  float *d_cudaB_distance;  
  hipMalloc(&d_cudaB_distance, n*n*sizeof(float));

  hipMemcpy(d_cudaB_distance, adjacency, n*n*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 threadsPerBlock; 
  dim3 blocksPerGrid;
  // Max number of threadsPerBlock is 1024 (32*32=1024)
  threadsPerBlock.x = threadsPerBlock.y = 32;
  blocksPerGrid.x = blocksPerGrid.y = n/32;

  int k;
  for (k = 0; k < n; ++k)
  {
    cudaB_kernel<<<blocksPerGrid, threadsPerBlock, 
      threadsPerBlock.x*sizeof(float)>>>(d_cudaB_distance, n, k);
  }

  hipMemcpy(cudaB_distance, d_cudaB_distance, n*n*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaC(float *adjacency, float *cudaC_distance, int n, float p, int w)
{
  float *d_cudaC_distance;  
  hipMalloc(&d_cudaC_distance, n*n*sizeof(float));

  int cellsPerThread = 8;

  hipMemcpy(d_cudaC_distance, adjacency, n*n*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 threadsPerBlock; 
  dim3 blocksPerGrid;
  // Max number of threadsPerBlock is 1024 (32*32=1024)  
  if (n/cellsPerThread < 32)
  {
    threadsPerBlock.x = threadsPerBlock.y = n/cellsPerThread;
    blocksPerGrid.x = 1;
  }
  else
  {
    threadsPerBlock.x = threadsPerBlock.y = 32;
    blocksPerGrid.x = blocksPerGrid.y = n/(32*cellsPerThread);
  }

  int k;
  for (k = 0; k < n; ++k)
  {
    cudaC_kernel<<<blocksPerGrid, threadsPerBlock, 
      (threadsPerBlock.x+threadsPerBlock.y)*cellsPerThread*sizeof(float)
        >>>(d_cudaC_distance, n, k, cellsPerThread);
  }

  hipMemcpy(cudaC_distance, d_cudaC_distance, n*n*sizeof(float), 
    hipMemcpyDeviceToHost);
}

__global__ void cudaA_kernel(float *d_cudaA_distance, int n, int k)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x*n+y < n*n)  
    if (d_cudaA_distance[x*n+y] > d_cudaA_distance[x*n+k] + d_cudaA_distance[k*n+y])
      d_cudaA_distance[x*n+y] = d_cudaA_distance[x*n+k] + d_cudaA_distance[k*n+y];
}

__global__ void cudaB_kernel(float *d_cudaB_distance, int n, int k)
{
  extern __shared__ float temp[];

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  temp[threadIdx.x] = d_cudaB_distance[x*n + k];

  if (x*n+y < n*n)  
    if (d_cudaB_distance[x*n+y] > temp[threadIdx.x] + d_cudaB_distance[k*n+y])
      d_cudaB_distance[x*n+y] = temp[threadIdx.x] + d_cudaB_distance[k*n+y];
}

__global__ void cudaC_kernel(float *d_cudaC_distance, int n, int k, int cellsPerThread)
{
  extern __shared__ float temp[];
  float *temp2 = &temp[blockDim.x*cellsPerThread];

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.x + threadIdx.y;  

  int i;  
  for (i=0; i<cellsPerThread; i++) 
  {
    temp[threadIdx.x*cellsPerThread + i] = d_cudaC_distance[(i+x*cellsPerThread)*n + k];
    temp2[threadIdx.y*cellsPerThread + i] = d_cudaC_distance[k*n + i + y*cellsPerThread];
  }

  __syncthreads();
  
  int j;  
  for (i=0; i<cellsPerThread; i++) 
    for (j=0; j<cellsPerThread; j++)
      if ( d_cudaC_distance[(i+x*cellsPerThread)*n + j + y*cellsPerThread] > 
        temp[threadIdx.x*cellsPerThread + i] + temp2[threadIdx.y*cellsPerThread + j] )
      {
        d_cudaC_distance[(i+x*cellsPerThread)*n + j + y*cellsPerThread] = 
          temp[threadIdx.x*cellsPerThread + i] + temp2[threadIdx.y*cellsPerThread + j];
      }
}

int compare(float *arrayA, float *arrayB, int n)
{
  for (int i = 0; i < n*n; ++i)
  {
    if (arrayB[i] != arrayA[i])
      return 0;    
  }
  return 1;
}



