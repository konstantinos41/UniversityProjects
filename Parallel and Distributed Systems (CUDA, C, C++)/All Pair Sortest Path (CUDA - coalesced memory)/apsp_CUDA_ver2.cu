/*
  mavrkons_apsp.cu

  All Pair Shortest Path version 2 (APSP)

  Project D

  Author: <Konstantinos Mavrodis>
  Contact: <kmavrodis@outlook.com>
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

void initialize(int *n, float *p, int *w, int argc, char const **argv);
void makeAdjacencies(float *adjacency, float *adjacencyT, int n, float p, int w);
void serialApsp(float *adjacency, float *distance, int n, float p , int w);
void cudaA(float *adjacency, float *adjacencyT, float *cudaA_distance,  int n, float p, int w);
void cudaB(float *adjacency, float *adjacencyT,float *cudaB_distance, int n, float p, int w);
void cudaC(float *adjacency, float *adjacencyT, float *cudaC_distance, int n, float p, int w);
__global__ void cudaA_kernel(float *d_cudaA_distance, float *d_cudaA_distanceT, int n, int k);
__global__ void cudaB_kernel(float *d_cudaB_distance, float *d_cudaB_distanceT, int n, int k);
__global__ void cudaC_kernel(float *d_cudaC_distance, float *d_cudaC_distanceT, int n, int k, int cellsPerThread);
int compare(float *arrayA, float *arrayB, int n);
void printMatrix (float *, int);


int main(int argc, char const **argv)
{
  int n, w;
  float p, *adjacency, *adjacencyT, *distance, *cudaA_distance, *cudaB_distance, *cudaC_distance;
  double serialTime, cudaATime, cudaBTime, cudaCTime;
  struct timespec tstart={0,0}, tend={0,0};

  initialize(&n, &p, &w, argc, argv);
  
  adjacency = (float*) malloc(n*n * sizeof(float));
  adjacencyT = (float*) malloc(n*n * sizeof(float));
  distance = (float*) malloc(n*n * sizeof(float));
  cudaA_distance = (float*) malloc(n*n * sizeof(float));
  cudaB_distance = (float*) malloc(n*n * sizeof(float));
  cudaC_distance = (float*) malloc(n*n * sizeof(float));

  makeAdjacencies(adjacency, adjacencyT, n, p, w);

  // Serial
  clock_gettime(CLOCK_MONOTONIC, &tstart);

  serialApsp(adjacency, distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  serialTime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) -
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);


  // Cuda A
  clock_gettime(CLOCK_MONOTONIC, &tstart);  

  cudaA(adjacency, adjacencyT, cudaA_distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  cudaATime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);


  // Cuda B
  clock_gettime(CLOCK_MONOTONIC, &tstart);  

  cudaB(adjacency, adjacencyT, cudaB_distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  cudaBTime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);
  

  // Cuda C
  clock_gettime(CLOCK_MONOTONIC, &tstart);  

  cudaC(adjacency, adjacencyT, cudaC_distance, n, p, w);

  clock_gettime(CLOCK_MONOTONIC, &tend);
  cudaCTime = ((double)tend.tv_sec + 1.0e-9*tend.tv_nsec) - 
    ((double)tstart.tv_sec + 1.0e-9*tstart.tv_nsec);


  printf("n=%i p=%g w=%i\n", n, p, w);
  printf("Serial time: %f\n", serialTime);
  printf("Cuda A time: %f\n", cudaATime);
  printf("Cuda B time: %f\n", cudaBTime);
  printf("Cuda C time: %f\n", cudaCTime);


  if(compare(cudaA_distance, distance,n) && compare(cudaB_distance, distance,n)
    && (compare(cudaC_distance, distance,n)))
    printf("PASS: All algorithms ran correctly.\n");
  else
    printf("FAIL: There was a wrong result.\n");

}


void initialize(int *n, float *p, int *w, int argc, char const **argv)
{
  if (argc == 4)
  {
    *n = 1<<atoi(argv[1]);
    *p = atof(argv[2]);
    *w = atoi(argv[3]); 
  }
  else
  {
    printf("The defaults for n,p,w (7,0.7,30) will be used\n");
    *n = 128;
    *p = 0.7;
    *w = 30;
  }
}

void makeAdjacencies(float *adjacency, float *adjacencyT, int n, float p , int w)
{
  int i, j;
  time_t t;
  srand((unsigned) time(&t));

  for (i = 0; i < n*n; ++i)
    adjacency[i] = 0;


  for (i = 0; i < n; ++i)
  {
    for (j = 0; j < n; ++j)
    {
      if ((float)(rand()%1000)/1000 > p)
        adjacency[i*n+j] = INFINITY;
      else
        adjacency[i*n+j] = ((float)(rand()%1000)/1000) * w;
    }
    adjacency[i*n+i] = 0;
  }

  // Transpose adjacency and store to adjacencyT
  for (int i = 0; i < n; ++i)
  {
  	for (int j = 0; j < n; ++j)
  	{
  		adjacencyT[i*n+j] = adjacency[j*n+i];
  	}
  }
}

void serialApsp(float *adjacency, float *distance, int n, float p , int w)
{
  int i, j, k;

  for (i = 0; i < n; ++i)
    for (j = 0; j < n; ++j)
      distance[i*n+j] = adjacency[i*n+j];

  for (k = 0; k < n; ++k)
    for (i = 0; i < n; ++i)
      for (j = 0; j < n; ++j)
        if (distance[i*n+j] > distance[i*n+k] + distance[k*n+j])
          distance[i*n+j] = distance[i*n+k] + distance[k*n+j];
}



void cudaA(float *adjacency, float *adjacencyT, float *cudaA_distance, int n, float p, int w)
{
  float *d_cudaA_distance;
  float *d_cudaA_distanceT;

  hipMalloc(&d_cudaA_distance, n*n*sizeof(float));
  hipMalloc(&d_cudaA_distanceT, n*n*sizeof(float));

  hipMemcpy(d_cudaA_distance, adjacency, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_cudaA_distanceT, adjacencyT, n*n*sizeof(float), hipMemcpyHostToDevice);

  
  dim3 threadsPerBlock; 
  dim3 blocksPerGrid;

  threadsPerBlock.x = threadsPerBlock.y = 32;
  blocksPerGrid.x = blocksPerGrid.y = n/32;

  int k;
  for (k = 0; k < n; ++k)
    cudaA_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_cudaA_distance, d_cudaA_distanceT, n, k);

  hipMemcpy(cudaA_distance, d_cudaA_distance, n*n*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaB(float *adjacency, float *adjacencyT, float *cudaB_distance, int n, float p, int w)
{
  float *d_cudaB_distance;
  float *d_cudaB_distanceT;  

  hipMalloc(&d_cudaB_distance, n*n*sizeof(float));
  hipMalloc(&d_cudaB_distanceT, n*n*sizeof(float));

  hipMemcpy(d_cudaB_distance, adjacency, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_cudaB_distanceT, adjacencyT, n*n*sizeof(float), hipMemcpyHostToDevice);
  
  dim3 threadsPerBlock; 
  dim3 blocksPerGrid;
  // Max number of threadsPerBlock is 1024 (32*32=1024)
  threadsPerBlock.x = threadsPerBlock.y = 32;
  blocksPerGrid.x = blocksPerGrid.y = n/32;

  int k;
  for (k = 0; k < n; ++k)
  {
    cudaB_kernel<<<blocksPerGrid, threadsPerBlock, 
      threadsPerBlock.x*sizeof(float)>>>(d_cudaB_distance, d_cudaB_distanceT, n, k);
  }

  hipMemcpy(cudaB_distance, d_cudaB_distance, n*n*sizeof(float), hipMemcpyDeviceToHost);
}

void cudaC(float *adjacency, float *adjacencyT, float *cudaC_distance, int n, float p, int w)
{
  float *d_cudaC_distance;
  float *d_cudaC_distanceT;  

  hipMalloc(&d_cudaC_distance, n*n*sizeof(float));
  hipMalloc(&d_cudaC_distanceT, n*n*sizeof(float));


  int cellsPerThread = 8;

  hipMemcpy(d_cudaC_distance, adjacency, n*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_cudaC_distanceT, adjacencyT, n*n*sizeof(float), hipMemcpyHostToDevice);

  dim3 threadsPerBlock; 
  dim3 blocksPerGrid;
  // Max number of threadsPerBlock is 1024 (32*32=1024)  
  if (n/cellsPerThread < 32)
  {
    threadsPerBlock.x = threadsPerBlock.y = n/cellsPerThread;
    blocksPerGrid.x = 1;
  }
  else
  {
    threadsPerBlock.x = threadsPerBlock.y = 32;
    blocksPerGrid.x = blocksPerGrid.y = n/(32*cellsPerThread);
  }

  int k;
  for (k = 0; k < n; ++k)
  {
    cudaC_kernel<<<blocksPerGrid, threadsPerBlock, 
      (threadsPerBlock.x+threadsPerBlock.y)*cellsPerThread*sizeof(float)
        >>>(d_cudaC_distance, d_cudaC_distanceT, n, k, cellsPerThread);
  }

  hipMemcpy(cudaC_distance, d_cudaC_distance, n*n*sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void cudaA_kernel(float *d_cudaA_distance, float *d_cudaA_distanceT, int n, int k)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x*n+y < n*n && x+y*n < n*n)  
    if (d_cudaA_distance[x*n+y] > d_cudaA_distanceT[k*n+x] + d_cudaA_distance[k*n+y])
    {
      d_cudaA_distance[x*n+y] = d_cudaA_distanceT[y*n+x] = d_cudaA_distanceT[k*n+x] + d_cudaA_distance[k*n+y];
    }
}


__global__ void cudaB_kernel(float *d_cudaB_distance, float *d_cudaB_distanceT, int n, int k)
{
  extern __shared__ float temp[];

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  temp[threadIdx.x] = d_cudaB_distanceT[k*n + x];

  if (x*n+y < n*n)  
      if (d_cudaB_distance[x*n+y] > temp[threadIdx.x] + d_cudaB_distance[k*n+y])
        d_cudaB_distance[x*n+y] = d_cudaB_distanceT[y*n+x] = temp[threadIdx.x] + d_cudaB_distance[k*n+y];
}


__global__ void cudaC_kernel(float *d_cudaC_distance, float *d_cudaC_distanceT, int n, int k, int cellsPerThread)
{
  extern __shared__ float temp[];
  float *temp2 = &temp[blockDim.x*cellsPerThread];

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.x + threadIdx.y;  

  int i;  
  for (i=0; i<cellsPerThread; i++) 
  {
    temp[threadIdx.x*cellsPerThread + i] = d_cudaC_distanceT[k*n + (i+x*cellsPerThread)];
    temp2[threadIdx.y*cellsPerThread + i] = d_cudaC_distance[k*n + i + y*cellsPerThread];
  }

  __syncthreads();
  
  int j;  
  for (i=0; i<cellsPerThread; i++) 
    for (j=0; j<cellsPerThread; j++)
      if ( d_cudaC_distance[(i+x*cellsPerThread)*n + j + y*cellsPerThread] > 
        temp[threadIdx.x*cellsPerThread + i] + temp2[threadIdx.y*cellsPerThread + j] )
      {
        d_cudaC_distance[(i+x*cellsPerThread)*n + j + y*cellsPerThread] = 
        d_cudaC_distanceT[(j + y*cellsPerThread)*n + (i+x*cellsPerThread)] = 
        temp[threadIdx.x*cellsPerThread + i] + temp2[threadIdx.y*cellsPerThread + j];
      }
}


int compare(float *arrayA, float *arrayB, int n)
{
  for (int i = 0; i < n*n; ++i)
  {
    if (arrayB[i] != arrayA[i])
      return 0;    
  }
  return 1;
}

void printMatrix(float *array, int n)
{
  for (int i = 0; i < n; ++i)
  {
    for (int j = 0; j < n; ++j)
    {
      printf("%g ", array[i*n+j]);
    }
    printf("\n");
  }
  printf("\n\n");
}
